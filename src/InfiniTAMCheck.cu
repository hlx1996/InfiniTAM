#include "hip/hip_runtime.h"
//
// Created by tommy on 1/31/19.
//

#include "../include/InfiniTAMCheck.h"
#include "../include/CLIEngine.h"
#include "../libs/ITMLib/ITMLibDefines.h"
#include "../libs/ITMLib/Objects/Scene/ITMScene.h"
#include "../libs/ITMLib/Core/ITMBasicEngine.h"
#include "../libs/ITMLib/Objects/Scene/ITMRepresentationAccess.h"

template<class TVoxel=ITMVoxel>
__device__ void InfiniTAMCheck_device(ITMLib::ITMScene<TVoxel, ITMVoxelIndex> *scene,
                                      ITMHashEntry *hashTable, TVoxel *localVBA,
                                      float x, float y, float z,
                                      float voxelSize, float requireSize, float *answer) {

    Vector3f point(x, y, z);
    int times = int(requireSize / voxelSize);

    int require_block_size = SDF_BLOCK_SIZE / times;
    float oneOverVoxelSizeBlockSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);
    float oneOverRequireSize = 1.0f / requireSize;

    Vector3s blockPos = TO_SHORT_FLOOR3(point * oneOverVoxelSizeBlockSize);
    Vector3s posInBlock =
            (TO_SHORT_FLOOR3(point * oneOverRequireSize) - blockPos * require_block_size) * times;
    int hashIdx = hashIndex(blockPos);

    ITMHashEntry hashEntry = hashTable[hashIdx];
    bool isFound = false;
    if (IS_EQUAL3(hashEntry.pos, blockPos) && hashEntry.ptr >= -1) {
        isFound = true;
        TVoxel *localVoxelBlock = &(localVBA[hashEntry.ptr * (SDF_BLOCK_SIZE3)]);
        float ans = 1e100;
        for (int i = 0; i < times; i++)
            for (int j = 0; j < times; j++)
                for (int k = 0; k < times; k++) {
                    float tmp = TVoxel::valueToFloat(
                            localVoxelBlock[(posInBlock[0] + i) + (posInBlock[1] + j) * SDF_BLOCK_SIZE +
                                            (posInBlock[2] + k) * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE].sdf);
                    if (fabs(tmp) < fabs(ans)) ans = tmp;
                }

        *answer = ans;
        return;
    }

    if (!isFound) {
        if (hashEntry.ptr >= -1) //search excess list only if there is no room in ordered part
        {
            while (hashEntry.offset >= 1) {
                hashIdx = SDF_BUCKET_NUM + hashEntry.offset - 1;
                hashEntry = hashTable[hashIdx];

                if (IS_EQUAL3(hashEntry.pos, blockPos) && hashEntry.ptr >= -1) {
                    TVoxel *localVoxelBlock = &(localVBA[hashEntry.ptr * (SDF_BLOCK_SIZE3)]);
                    float ans = 1e100;
                    for (int i = 0; i < times; i++)
                        for (int j = 0; j < times; j++)
                            for (int k = 0; k < times; k++) {
                                float tmp = TVoxel::valueToFloat(localVoxelBlock[(posInBlock[0] + i)
                                                                                 + (posInBlock[1] + j) * SDF_BLOCK_SIZE
                                                                                 +
                                                                                 (posInBlock[2] + k) * SDF_BLOCK_SIZE *
                                                                                 SDF_BLOCK_SIZE].sdf);
                                if (fabs(tmp) < fabs(ans)) ans = tmp;
                            }
                    *answer = ans;
                    return;
                }
            }
        }
    }
    *answer = 1e100;
//    return TVoxel();
}

template<class TVoxel=ITMVoxel>
__global__ void InfiniTAMCheck_global(ITMLib::ITMScene<TVoxel, ITMVoxelIndex> *scene,
                                      ITMHashEntry *hashTable, TVoxel *localVBA,
                                      float x, float y, float z,
                                      float voxelSize, float requireSize, float *answer) {
    InfiniTAMCheck_device(scene, hashTable, localVBA, x, y, z, voxelSize, requireSize, answer);

}

float InfiniTAMCheck(ITMLib::ITMMainEngine *mainEngine, float x, float y, float z, float requireSize) {
    ITMLib::ITMScene<ITMVoxel_s, ITMVoxelIndex> *scene
            = dynamic_cast<ITMLib::ITMBasicEngine<ITMVoxel_s, ITMVoxelIndex> *>(mainEngine)->getScene();
    ITMHashEntry *hashTable = scene->index.GetEntries();

    ITMVoxel_s *localVBA = scene->localVBA.GetVoxelBlocks();
    float *answer_device, answer_host;
    hipMalloc(&answer_device, sizeof(float));
    InfiniTAMCheck_global << < 1, 1 >> >
                                  (scene, hashTable, localVBA, x, y, z,
                                          scene->sceneParams->voxelSize, requireSize, answer_device);
    hipMemcpy(&answer_host, answer_device, sizeof(float), hipMemcpyDeviceToHost);
    ORcudaSafeCall(hipDeviceSynchronize());
    if (answer_host > 1 || answer_host < -1) return 1e100;
    if (answer_host > 0.9999 || answer_host < -0.9999) return 100;
    return answer_host * scene->sceneParams->mu;
}
